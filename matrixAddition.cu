#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <iostream>

using namespace std;

#define N 10 // rows
#define M 10 // columns
#define Num_threads_x 2
#define Num_threads_y 4
#define Num_elements 100

__global__ void addMatrixCUDA(const int *a, const int *b, int *c)
{
        int x = blockIdx.x * blockDim.x + threadIdx.x;
        int y = blockIdx.y * blockDim.y + threadIdx.y;
        int i = y * N + x;
        if (i < Num_elements)
                c[i] = a[i] + b[i];

}

void addMatrix(int *a, int *b, int *c)
{
        int *dev_a;
        int *dev_b;
        int *dev_c;

        dim3 blocks(Num_elements/Num_threads_x, Num_elements/Num_threads_y);
        dim3 threads(Num_threads_x, Num_threads_y);

        hipMalloc((void**)&dev_c, Num_elements * sizeof(int));
        hipMalloc((void**)&dev_a, Num_elements * sizeof(int));
        hipMalloc((void**)&dev_b, Num_elements * sizeof(int));

        hipMemcpy(dev_a, a, Num_elements * sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(dev_b, b, Num_elements * sizeof(int), hipMemcpyHostToDevice);

        addMatrixCUDA << <blocks, threads >> > (dev_a, dev_b, dev_c);

        hipDeviceSynchronize();

        hipMemcpy(c, dev_c, Num_elements * sizeof(int), hipMemcpyDeviceToHost);
}

int main()
{
        int a[Num_elements], b[Num_elements], c[Num_elements];
        for (int i = 0; i < Num_elements; i++)
        {
                a[i] = -i;
                b[i] = i * i;
        }
        addMatrix(a, b, c);
        for (int i = 0; i < N; i++)
        {
                for (int j = 0; j < M; j++)
                {
                        int idx = i * N + j;
                        cout << c[idx] << '\t';
                }
                cout << endl;
        }

    return 0;
}

