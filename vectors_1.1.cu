#include "hip/hip_runtime.h"

#include <stdio.h>

#define N 10

__global__ void addVect(int *a, int *b, int *c)
{
        //for <<<1, N>>>
        //int i = threadIdx.x;

        //for <<<N,1>>>
        int i = blockIdx.x;

        //for <<<1, 1>>>
        //for (int i = 0; i < N; i++)
                c[i] = a[i] + b[i];
}

void add(int *a, int *b, int *c)
{
	int *dev_a;
	int *dev_b;
	int *dev_c;

	hipMalloc((void**)&dev_c, N * sizeof(int));
	hipMalloc((void**)&dev_a, N * sizeof(int));
	hipMalloc((void**)&dev_b, N * sizeof(int));

	hipMemcpy(dev_a, a, N * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_b, b, N * sizeof(int), hipMemcpyHostToDevice);

	//addVect <<<1, N >>> (dev_a, dev_b, dev_c);
	//addVect <<<1, 1>>> (dev_a, dev_b, dev_c);
	addVect <<<N, 1 >>> (dev_a, dev_b, dev_c);

	hipDeviceSynchronize();
	hipMemcpy(c, dev_c, N * sizeof(int), hipMemcpyDeviceToHost);

}
int main()
{
	int a[N], b[N], c[N];
	for (int i = 0; i < N; i++)
	{
		a[i] = -i;
		b[i] = i * i;
	}

	add(a, b, c);

	// display the results
	for (int i = 0; i < N; i++)
	{
		printf("%d + %d = %d\n", a[i], b[i], c[i]);
	}

	return 0;
}
