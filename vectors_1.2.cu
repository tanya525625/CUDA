#include "hip/hip_runtime.h"

#include <stdio.h>

#define N 100
#define num_threads 10

__global__ void addVect(int *a, int *b, int *c)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	c[i] = a[i] + b[i];
}

void add(int *a, int *b, int *c)
{
	int *dev_a;
	int *dev_b;
	int *dev_c;

	hipMalloc((void**)&dev_c, N * sizeof(int));
	hipMalloc((void**)&dev_a, N * sizeof(int));
	hipMalloc((void**)&dev_b, N * sizeof(int));

	hipMemcpy(dev_a, a, N * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_b, b, N * sizeof(int), hipMemcpyHostToDevice);

	int num_blocks = (N + num_threads - 1) / num_threads;
	addVect <<< num_blocks, num_threads >>> (dev_a, dev_b, dev_c);

	hipDeviceSynchronize();
	hipMemcpy(c, dev_c, N * sizeof(int), hipMemcpyDeviceToHost);

}
int main()
{
	int a[N], b[N], c[N];
	for (int i = 0; i < N; i++)
	{
		a[i] = -i;
		b[i] = i * i;
	}

	add(a, b, c);

	// display the results
	for (int i = 0; i < N; i++)
	{
		printf("%d + %d = %d\n", a[i], b[i], c[i]);
	}

	return 0;
}
